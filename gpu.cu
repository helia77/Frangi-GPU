#include "hip/hip_runtime.h"
﻿#include<iostream>
#include"hip/hip_runtime.h"
#include<hip/hip_runtime_api.h>
#include""
#include"tira/image.h"
#include<tira/image/colormap.h>
//#include"frangi.h"

#define _USE_MATH_DEFINES
#define __HIPCC__
# define PI 3.14159265358979323846  /* pi */

static void HandleError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << "in" << file << "at line" << line;
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ )) 


//  convolution on device
__global__ void dev_conv(float* out, float* img, float* kernel, int img_w, int out_h, int out_w, int K) {
    size_t i = blockDim.y * blockIdx.y + threadIdx.y;
    size_t j = blockDim.x * blockIdx.x + threadIdx.x;
    
    // i and j being smaller than output's width and height, manage the edges perfectly
    if (i >= out_h || j >= out_w) return;

    float conv = 0;
    for (int ki = 0; ki < K; ki++)
        for (int kj = 0; kj < K; kj++)
            conv += img[(i + ki) * img_w + j + kj] * kernel[ki*K + kj];
        
    out[i * out_w + j] = conv;

}

// convolving a kernel with an image using GPU
tira::image<float> convolution_gpu(tira::image<float>& img, float* kernel, int k_size) {

    tira::image<float> src(img);
    int size = src.width() * src.height();		        // size of the image
    
    // output sizes after convolution
    int y_height = src.height() - k_size + 1;
    int y_width = src.width() - k_size + 1;
    int y_size = y_height * y_width;
    float* y_output = (float*)malloc(y_size * sizeof(float));

    // -------------------------------------- GPU ---------------------------------------- //
    int d;
    HANDLE_ERROR(hipGetDevice(&d));
    //std::cout << "Current device: " << d << std::endl;
    hipDeviceProp_t prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, d));
    
    float* gpu_gKernel;
    float* gpu_image;
    float* gpu_output_x;
    float* gpu_output_y;
    
    // allocate memory for image, kernel, and convoled output
    HANDLE_ERROR(hipMalloc(&gpu_gKernel, k_size * k_size * sizeof(float)));
    HANDLE_ERROR(hipMalloc(&gpu_image, size * sizeof(float)));
    HANDLE_ERROR(hipMalloc(&gpu_output_y, y_size * sizeof(float)));

    float* imageArr = img.data();
    float* gKernel = kernel;

    // copy image and kernel from main memory to Device
    HANDLE_ERROR(hipMemcpy(gpu_image, imageArr, size * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(gpu_gKernel, gKernel, k_size * k_size * sizeof(float), hipMemcpyHostToDevice));


    size_t blockDim = sqrt(prop.maxThreadsPerBlock);
    dim3 threads(blockDim, blockDim);
    //std::cout << "w: " << width << std::endl << "h: " << height << std::endl;
    dim3 blocks(src.width() / threads.x + 1, src.height() / threads.y + 1);

    // starting GPU timer
    /*hipEvent_t g_start;
    hipEvent_t g_stop;
    hipEventCreate(&g_start);
    hipEventCreate(&g_stop);
    hipEventRecord(g_start, NULL);*/

    // convolving
    dev_conv << < blocks, threads >> > (gpu_output_y, gpu_image, gpu_gKernel, src.width(), y_height, y_width, k_size);
    
    // copy back the results to main memory
    HANDLE_ERROR(hipMemcpy(y_output, gpu_output_y, y_size * sizeof(float), hipMemcpyDeviceToHost));

    // GPU timer ends
    /*hipEventRecord(g_stop, NULL);
    hipEventSynchronize(g_stop);
    float eTime;
    hipEventElapsedTime(&eTime, g_start, g_stop);
    std::cout << "Takes " << eTime << " ms to convolve on GPU" << std::endl;*/

    tira::image<float> output(y_output, y_width, y_height);
    
    return output;
}